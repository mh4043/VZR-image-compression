#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// STB
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../packages/stb_image.h"
#include "../packages/stb_image_write.h"

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../packages/hip/hip_runtime_api.h"

int width, height, cpp;  // Image qualities
unsigned char *image_original;

// Function parameters
char *image_file;
int number_of_clusters, number_of_iterations;

#define BINS 256
#define UNIFIED_MEMORY
#define THREADS_PER_BLOCK 256

__device__ int get_closest_cluster(unsigned char *pixel, int number_of_clusters, unsigned int *clusters, int cpp) {
    // Iterate number of clusters, check which one is closest in terms of euclidean distance of RGB
    float min_dst = 1000;  // minimum distance from pixel to centroid
    int min_indx = 0;      // centroid index with min distance to pixel
    for (int centroid = 0; centroid < number_of_clusters; centroid++) {
        float dst = 0.0;

        // Iterate cpp to get RGB values, accumulate distance
        for (int j = 0; j < cpp; j++) {
            char cntr = clusters[centroid * cpp + j];
            char delta = pixel[j] - cntr;
            dst += powf(delta, 2);
        }

        // Get euclidean distance
        dst = sqrtf(dst);

        // Check if distance is smaller than min_dst
        if (dst < min_dst) {
            min_dst = dst;
            min_indx = centroid;
        }
    }
    return min_indx;
}

__global__ void fill_closest_clusters(
    unsigned char *image,
    unsigned int *clusters,
    int *cluster_assignments,
    int number_of_clusters,
    int width,
    int height,
    int cpp) {

    // Get thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Get pixel

    unsigned char *pixel = &image[tid * cpp];

    // Assign pixel to cluster
    cluster_assignments[tid] = get_closest_cluster(pixel, number_of_clusters, clusters, cpp);
}

__global__ void update_centroids(unsigned char *image,
                                 unsigned int *clusters,
                                 int *cluster_assignments,
                                 int number_of_clusters,
                                 int width, int height, int cpp) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int block_tid = threadIdx.x;
    
    __shared__ unsigned char shared_image[THREADS_PER_BLOCK * 4];
    // Create shared memory
    if (cpp == 3)
        __shared__ unsigned char shared_image[THREADS_PER_BLOCK * 3];
    
    for (int j = 0; j < cpp; j++) 
        shared_image[block_tid *cpp + j] = image[tid * cpp + j];

    // Save cluster assignments to shared memory
    __shared__ int cluster_assignments_shared[THREADS_PER_BLOCK];
    cluster_assignments_shared[block_tid] = cluster_assignments[tid];

    // Wait for all threads to finish
    __syncthreads();

    // If block_tid is 0, then we are in the first thread of the block, so we can do the reduction
    if (block_tid == 0) {
        // Cluster counts - count how many pixels have cluster[i] as closest, store in shared memory, define size as number_of_clusters
        int cluster_counts_shared[THREADS_PER_BLOCK] = {0};

        // Cluster centroids - sum all pixels that have cluster[i] as closest, store in shared memory, define size as number_of_clusters * cpp
        unsigned int clusters_shared[THREADS_PER_BLOCK * 4] = {0};
        if (cpp == 3)
            unsigned int clusters_shared[THREADS_PER_BLOCK * 3] = {0};


        // Iterate over all threads
        for (int i = 0; i < THREADS_PER_BLOCK; i++) {
            // Get cluster assignment
            int cluster_assignment = cluster_assignments_shared[i];
            // Increment cluster count
            cluster_counts_shared[cluster_assignment] += 1;
            // Increment cluster centroid
            for (int j = 0; j < cpp; j++) 
                clusters_shared[cluster_assignment * cpp + j] += shared_image[i *cpp + j];
        }

        // Save cluster counts to global memory
        for (int i = 0; i < number_of_clusters; i++) {
            cluster_assignments[i] = cluster_counts_shared[i];
        }
    }
    // Wait for all threads to finish
    __syncthreads();

    if (tid < number_of_clusters) {
        clusters[tid] /= cluster_assignments[tid];
    }
}

__global__ void save_images(unsigned char *image, unsigned int *clusters, int *cluster_assignments, int cpp) {
    // Get thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Save image
    for (int j = 0; j < cpp; j++) {
        image[tid * cpp + j] = clusters[cluster_assignments[tid] * cpp + j];
    }
}

int main(int argc, char *argv[]) {
    srand(time(NULL));

    if (argc > 3) {
        image_file = argv[1];
        number_of_clusters = atoi(argv[2]);
        number_of_iterations = atoi(argv[3]);
    } else {
        fprintf(stderr, "Not enough arguments\n");
        fprintf(stderr, "Usage: %s <IMAGE_PATH> <number of colors> <number of iterations>\n", argv[0]);
        exit(1);
    }

    fflush(stdout);
    image_original = stbi_load(image_file, &width, &height, &cpp, 0);

    if (image_original) {
        // Define Thread number
        #define SIZE (width * height)

        // Thread organization
        dim3 blockSize(THREADS_PER_BLOCK);
        dim3 gridSize((SIZE + blockSize.x - 1) / blockSize.x);

        printf("Image size: %d, Image width: %d, Image height: %d, Image channels: %d, Number of clusters: %d, Number of iterations: %d\n", SIZE, width, height, cpp, number_of_clusters, number_of_iterations);
        fflush(stdout);
        printf("Block size: %d, Grid size: %d\n", blockSize.x, gridSize.x);
        fflush(stdout);

        // Init clusters as 1D array
        unsigned int clusters_centroids[number_of_clusters * cpp];  // This will map cluster to pixel, each cluster will have a pixel
        for (int i = 0; i < number_of_clusters; i++) {
            int random_pixel_w = (i * width) / number_of_clusters;
            int random_pixel_h = (i * height) / number_of_clusters;

            for (int j = 0; j < cpp; j++) {
                clusters_centroids[i * cpp + j] = image_original[(random_pixel_h * width + random_pixel_w) * cpp + j];
            }
        }

        // Init cluster assignments, use malloc
        int *cluster_assignments = (int *)malloc(SIZE * sizeof(int));

        // Init cluster counts
        int cluster_counts[number_of_clusters];  // This will map cluster to number of pixels

        // Copy image to device
        unsigned char *image_device;
        checkCudaErrors(hipMalloc(&image_device, SIZE * cpp * sizeof(unsigned char)));
        checkCudaErrors(hipMemcpy(image_device, image_original, SIZE * cpp * sizeof(unsigned char), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy image_original failed");

        // Copy clusters to device as 1D array
        unsigned int *clusters_device;
        checkCudaErrors(hipMalloc(&clusters_device, number_of_clusters * cpp * sizeof(unsigned int)));
        checkCudaErrors(hipMemcpy(clusters_device, clusters_centroids, number_of_clusters * cpp * sizeof(unsigned int), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy clusters_centroids failed");

        // Copy cluster counts to device
        int *cluster_counts_device;
        checkCudaErrors(hipMalloc(&cluster_counts_device, number_of_clusters * sizeof(int)));
        checkCudaErrors(hipMemcpy(cluster_counts_device, cluster_counts, number_of_clusters * sizeof(int), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy cluster_counts failed");

        // Copy cluster assignments to device
        int *cluster_assignments_device;
        checkCudaErrors(hipMalloc(&cluster_assignments_device, SIZE * sizeof(int)));
        checkCudaErrors(hipMemcpy(cluster_assignments_device, cluster_assignments, SIZE * sizeof(int), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy cluster_assignments failed");

        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        for (int iter = 0; iter < number_of_iterations; iter++) {
            // Assign each pixel to a cluster
            fill_closest_clusters<<<gridSize, blockSize>>>(image_device,
                                                           clusters_device,
                                                           cluster_assignments_device,
                                                           number_of_clusters,
                                                           width,
                                                           height,
                                                           cpp);

            // Update centroids
            update_centroids<<<gridSize, blockSize>>>(image_device,
                                                      clusters_device,
                                                      cluster_assignments_device,
                                                      number_of_clusters,
                                                      width,
                                                      height,
                                                      cpp);

        }
        // Save image
        save_images<<<gridSize, blockSize>>>(image_device, clusters_device, cluster_assignments_device, cpp);

        getLastCudaError("kmeans_clustering failed");

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("GPU Time: %0.3f milliseconds \n", milliseconds);

        // Copy clusters to host
        checkCudaErrors(hipMemcpy(clusters_centroids, clusters_device, number_of_clusters * cpp * sizeof(unsigned int), hipMemcpyDeviceToHost));
        getLastCudaError("hipMemcpy clusters_centroids failed");

        // Copy cluster counts to host
        checkCudaErrors(hipMemcpy(cluster_counts, cluster_counts_device, number_of_clusters * sizeof(int), hipMemcpyDeviceToHost));
        getLastCudaError("hipMemcpy cluster_counts failed");

        // Copy image to host
        checkCudaErrors(hipMemcpy(image_original, image_device, SIZE * cpp * sizeof(unsigned char), hipMemcpyDeviceToHost));
        getLastCudaError("hipMemcpy image_original failed");

        if (cpp == 3) {
            // Check if the image was written
            if (stbi_write_jpg("compressed_image.jpg",
                               width, height, cpp, image_original, 100)) {
                printf("Image written successfully\n");
            } else {
                printf("Error writing image\n");
            }
        }
        if (cpp == 4) {
            int stride_in_bytes = width * cpp;
            // Check if the image was written
            if (stbi_write_png("compressed_image.png",
                               width, height, cpp, image_original, stride_in_bytes)) {
                printf("Image written successfully\n");
            } else {
                printf("Error writing image\n");
            }
        }

        // Free memory
        checkCudaErrors(hipFree(image_device));
        checkCudaErrors(hipFree(clusters_device));
        checkCudaErrors(hipFree(cluster_counts_device));

    } else {
        fprintf(stderr, "Error loading image %s!\n", image_file);
    }
}