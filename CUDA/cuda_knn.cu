#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// STB
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../packages/stb_image.h"
#include "../packages/stb_image_write.h"

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../packages/hip/hip_runtime_api.h"

int width, height, cpp;  // Image qualities
unsigned char *image_original;

// Function parameters
char *image_file;
int number_of_clusters, number_of_iterations;

#define BINS 256
#define UNIFIED_MEMORY
#define THREADS_PER_BLOCK (1024)

__global__ void kmeans_clustering(
    unsigned char *image,
    unsigned int *clusters,
    int *cluster_counts,
    int *cluster_assignments,
    int number_of_clusters,
    int width,
    int height,
    int cpp,
    int iterations) {
    // Get thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int iteration = 0; iteration < iterations; iteration++) {
        // Get pixel
        unsigned char *pixel = &image[tid * cpp];

        // Find closest cluster with euclidean distance
        int closest_cluster = -1;
        int closest_distance = 999999999;

        for (int i = 0; i < number_of_clusters; i++) {
            int distance = 0;
            for (int j = 0; j < cpp; j++) {
                distance += (pixel[j] - clusters[i * cpp + j]) * (pixel[j] - clusters[i * cpp + j]);
            }
            if (distance < closest_distance) {
                closest_distance = distance;
                closest_cluster = i;
            }
        }

        // Assign pixel to cluster
        cluster_assignments[tid] = closest_cluster;

        // Add pixel to cluster count
        atomicAdd(&cluster_counts[closest_cluster], 1);

        // Add pixel to cluster sum
        for (int i = 0; i < cpp; i++) {
            atomicAdd(&clusters[closest_cluster * cpp + i], pixel[i]);
        }

        // Sync threads
        __syncthreads();

        if (tid < number_of_clusters) {
            /*
            // Check if cluster is empty - add the next closest pixel
            if (cluster_counts[tid] == 0) {
                int closest_distance = 999999999;
                int closest_pixel = -1;
                for (int j = 0; j < width * height; j++) {
                    int distance = 0;
                    for (int k = 0; k < cpp; k++) {
                        distance += (image[j * cpp + k] - clusters[tid * cpp + k]) * (image[j * cpp + k] - clusters[tid * cpp + k]);
                    }
                    if (distance < closest_distance) {
                        closest_distance = distance;
                        closest_pixel = j;
                    }
                }
                for (int j = 0; j < cpp; j++) {
                    clusters[tid * cpp + j] = image[closest_pixel * cpp + j];
                }
            }
            */

            for (int j = 0; j < cpp; j++) {
                //if (cluster_counts[tid] == 0)
                    //printf("Cluster %d has no count\n", tid);
                clusters[tid * cpp + j] = clusters[tid * cpp + j] / cluster_counts[tid];
            }
        }
        __syncthreads();
    }

    // Sync threads
    __syncthreads();

    // Save image
    for (int j = 0; j < cpp; j++) {
        image[tid * cpp + j] = clusters[cluster_assignments[tid] * cpp + j];
    }
}

int main(int argc, char *argv[]) {
    srand(time(NULL));

    if (argc > 3) {
        image_file = argv[1];
        number_of_clusters = atoi(argv[2]);
        number_of_iterations = atoi(argv[3]);
    } else {
        fprintf(stderr, "Not enough arguments\n");
        fprintf(stderr, "Usage: %s <IMAGE_PATH> <number of colors> <number of iterations>\n", argv[0]);
        exit(1);
    }

    printf("Starting program with parameters %s, %d, %d\n", image_file, number_of_clusters, number_of_iterations);
    fflush(stdout);
    image_original = stbi_load(image_file, &width, &height, &cpp, 0);

    if (image_original) {
// Define Thread number
#define SIZE (width * height)

        // Thread organization
        dim3 blockSize(THREADS_PER_BLOCK);
        dim3 gridSize((SIZE) / blockSize.x);

        printf("Image loaded with dimensions: %d %d %d\n", width, height, cpp);
        fflush(stdout);

        // Init clusters as 1D array
        unsigned int clusters_centroids[number_of_clusters * cpp];  // This will map cluster to pixel, each cluster will have a pixel
        for (int i = 0; i < number_of_clusters; i++) {
            int random_pixel_w = (i * width) / number_of_clusters;
            int random_pixel_h = (i * height) / number_of_clusters;

            for (int j = 0; j < cpp; j++) {
                clusters_centroids[i * cpp + j] = image_original[(random_pixel_h * width + random_pixel_w) * cpp + j];
            }
            printf("X and Y for cluster: %d, %d, \n", random_pixel_w, random_pixel_h);
        }

        // Init cluster assignments, use malloc
        int *cluster_assignments = (int *)malloc(SIZE * sizeof(int));

        // Init cluster counts
        int cluster_counts[number_of_clusters];  // This will map cluster to number of pixels

        // Copy image to device
        unsigned char *image_device;
        checkCudaErrors(hipMalloc((void **)&image_device, SIZE * cpp * sizeof(unsigned char)));
        checkCudaErrors(hipMemcpy(image_device, image_original, SIZE * cpp * sizeof(unsigned char), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy image_original failed");

        // Copy clusters to device as 1D array
        unsigned int *clusters_device;
        checkCudaErrors(hipMalloc((void **)&clusters_device, number_of_clusters * cpp * sizeof(unsigned int)));
        checkCudaErrors(hipMemcpy(clusters_device, clusters_centroids, number_of_clusters * cpp * sizeof(unsigned char), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy clusters_centroids failed");

        // Copy cluster counts to device
        int *cluster_counts_device;
        checkCudaErrors(hipMalloc((void **)&cluster_counts_device, number_of_clusters * sizeof(int)));
        checkCudaErrors(hipMemcpy(cluster_counts_device, cluster_counts, number_of_clusters * sizeof(int), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy cluster_counts failed");

        // Copy cluster assignments to device
        int *cluster_assignments_device;
        checkCudaErrors(hipMalloc((void **)&cluster_assignments_device, SIZE * sizeof(unsigned int)));
        checkCudaErrors(hipMemcpy(cluster_assignments_device, cluster_assignments, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice));
        getLastCudaError("hipMemcpy cluster_assignments failed");

        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Assign each pixel to a cluster
        kmeans_clustering<<<gridSize, blockSize>>>(
            image_device,
            clusters_device,
            cluster_counts_device,
            cluster_assignments_device,
            number_of_clusters,
            width,
            height,
            cpp,
            number_of_iterations);
        getLastCudaError("kmeans_clustering failed");

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("GPU Time: %0.3f milliseconds \n", milliseconds);

        // Copy clusters to host
        checkCudaErrors(hipMemcpy(clusters_centroids, clusters_device, number_of_clusters * cpp * sizeof(unsigned int), hipMemcpyDeviceToHost));
        getLastCudaError("hipMemcpy clusters_centroids failed");

        // Copy cluster counts to host
        checkCudaErrors(hipMemcpy(cluster_counts, cluster_counts_device, number_of_clusters * sizeof(int), hipMemcpyDeviceToHost));
        getLastCudaError("hipMemcpy cluster_counts failed");

        // Copy image to host
        checkCudaErrors(hipMemcpy(image_original, image_device, SIZE * cpp * sizeof(unsigned char), hipMemcpyDeviceToHost));
        getLastCudaError("hipMemcpy image_original failed");

        char output[] = "output_compressed";
        if (cpp == 3) {
            strcat(output, ".jpg");
            printf("%s", output);
            // Check if the image was written
            if (stbi_write_jpg(output,
                               width, height, cpp, image_original, 100)) {
                printf("Image written successfully\n");
            } else {
                printf("Error writing image\n");
            }
        }
        if (cpp == 4) {
            strcat(output, ".png");
            printf("%s", output);
            int stride_in_bytes = width * cpp;
            // Check if the image was written
            if (stbi_write_png(output,
                               width, height, cpp, image_original, stride_in_bytes)) {
                printf("Image written successfully\n");
            } else {
                printf("Error writing image\n");
            }
        }

        // Free memory
        checkCudaErrors(hipFree(image_device));
        checkCudaErrors(hipFree(clusters_device));
        checkCudaErrors(hipFree(cluster_counts_device));

    } else {
        fprintf(stderr, "Error loading image %s!\n", image_file);
    }
}